/*
** Olivia Caulfield
** Cho
** CSC 347
** 3/16/23
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define BLOCK_DIM 16

__global__ void histMethod(int numDigits, int piArray[], int *numCount);

int main(int argc, char **argv){
    printf("Distribution of the Digits of Pi: CUDA Code\n");
    
    // create count array on host
    int numCount[] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    // create count array on device
    int* dev_numCount;
    hipMalloc((void**) &dev_numCount, 10*sizeof(int));
    // copy host array to device array
    hipMemcpy(dev_numCount, numCount, 10*sizeof(int), hipMemcpyHostToDevice);
 
    // error check for correct number of command line arguments
    if (argc != 3){
        printf("Invalid number of command line arguments.\n");
        exit(100);
    }
    // File pointer to digits of pi file for CPU memory
    FILE *piFile;
    piFile = fopen(argv[1], "r");
    if(piFile == NULL){
        printf("File '%s' not found\n", argv[1]);
        exit (101);
    }
    // error check for argument being a positive integer
    int numDigits = atoi(argv[2]); 
    if (numDigits <= 0){
        printf("'%s' is not a positive integer.\n", argv[2]);
        exit(102);
    }

    // make array in host memory to store numDigits of pi
    int* piArray = (int*) malloc(numDigits*sizeof(int));
    char c;
    for (int i = 0; i < numDigits; i++){
        c = fgetc(piFile);
        if(c == EOF){
            printf("End of file!\n");
            numDigits = i;
            break;
        }
        piArray[i] = ((int)c)-48;
    }

    // make array in device memory to store numDigits of pi
    int* dev_piArray;
    hipMalloc((void**) &dev_piArray, numDigits*sizeof(int));
    // copy host array to device array
    hipMemcpy(dev_piArray, piArray, numDigits*sizeof(int), hipMemcpyHostToDevice);

    int N = numDigits;
    // define grid and block size
    dim3 dimBlock(BLOCK_DIM); 
    dim3 dimGrid((int)ceil(N+1/BLOCK_DIM));

    // allocate timers
    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);
    // start timer
    hipEventRecord(start,0);

    // call monte carlo method kernel
    histMethod<<<dimGrid,dimBlock>>>(numDigits, dev_piArray, dev_numCount);
    //histMethod<<<1,N>>>(numDigits, dev_piArray, dev_numCount);
    hipDeviceSynchronize();

    // copy device count array to host array 
    hipMemcpy(numCount, dev_numCount, 10*sizeof(int), hipMemcpyDeviceToHost);

    // stop timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float diff;
    hipEventElapsedTime(&diff, start, stop);
    // deallocate timers
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("time spent: %f\n", diff);
    
    for (int i = 0; i < 10; i++){
        printf("%d: %d\n", i, numCount[i]);
    }

    // write time to file
    FILE *fp;
    fp = fopen("piCUDA.csv", "w");
    for (int i = 0; i < 10; i++){
        fprintf(fp, "%f\n", (double)numCount[i]/numDigits*100);
    }
    fclose(fp);
    fclose(piFile);
    
    free(piArray);
    hipFree(dev_piArray);
    hipFree(dev_numCount);
    exit(0);
}

__global__ void histMethod(int numDigits, int piArray[], int *numCount){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < numDigits){
        int num = piArray[i];
        atomicAdd(&numCount[num], 1);
        //printf("%d\n", numCount[num]);
        //printf("%d: %d\n", numCount[i], num);
        //atomicAdd(&numCount[num], 1);
    }
}