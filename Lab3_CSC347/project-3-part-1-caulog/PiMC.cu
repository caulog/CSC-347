/*
** Olivia Caulfield
** Cho
** CSC 347
** 3/16/23
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define BLOCK_DIM 16

__global__ void mcMethod(int iter, float *randPoints, int* inside);

// kernel to calculate random numbers
__global__ void generateRandomNumbers(hiprandState_t* states, float* numbers, int n);

int main(int argc, char **argv){
    printf("Monte Carlo Method: CUDA Code\n");

    //for (int reps = 0; reps < 100000000){
    // error check for correct number of command line arguments
    if (argc != 2){
        printf("Invalid number of command line arguments.\n");
        exit(100);
    }
    // error check for argument being a positive integer
    int iter = atoi(argv[1]); 
    if (iter <= 0){
        printf("'%s' is not a positive integer.\n", argv[1]);
        exit(101);
    }
    // store the total number of points being calculated
    int total = iter;
    
    // host count for points inside circle
    int inside[]= {0};
    // device count for points inside circle 
    int* dev_inside;
    hipMalloc((void**) &dev_inside, sizeof(int));
    // copy host array to device arry
    hipMemcpy(dev_inside, inside, sizeof(int), hipMemcpyHostToDevice);
    
    // define size of array
    int N = iter*2;

    // define grid and block size
    dim3 dimBlock(BLOCK_DIM); 
    dim3 dimGrid((int)ceil(N+1/BLOCK_DIM));

    // host array for points
    float *randPoints = (float*) malloc(N *sizeof(float));
    // initialize as zeros
    for (int i = 0; i < N; i++){ randPoints[i] = 0; }
    // device array for points
    float *dev_randPoints;
    hipMalloc((void**) &dev_randPoints, N *sizeof(float));
    // copy the host array of zeros to the device 
    hipMemcpy(dev_randPoints, randPoints, N*sizeof(float), hipMemcpyHostToDevice);

    // device memory for curandState_t objects
    hiprandState_t* states;
    hipMalloc(&states, N * sizeof(hiprandState_t));

    // generate random numers
    generateRandomNumbers<<<dimGrid, dimBlock>>>(states, dev_randPoints, N);
    hipDeviceSynchronize();
    // copy to host
    hipMemcpy(randPoints, dev_randPoints, N*sizeof(float), hipMemcpyDeviceToHost);

    /**error check
    for (int i = 0; i < iter; i++){
        printf("%d (%f, %f)\n", i, randPoints[i*2], randPoints[i*2+1]);
    }*/

    // allocate timers
    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);
    // start timer
    hipEventRecord(start,0);

    // call monte carlo method kernel
    mcMethod<<<dimGrid,dimBlock>>>(iter, dev_randPoints, dev_inside);
    hipDeviceSynchronize();
    // copy device to host
    hipMemcpy(inside, dev_inside, sizeof(int), hipMemcpyDeviceToHost);

    // stop timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float diff;
    hipEventElapsedTime(&diff, start, stop);
    // deallocate timers
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("time spent: %f\n", diff);
   
    // calculate pi
    int in = inside[0];
    float pi = float(4 * in )/total;
    printf("pi estimated: %f\n", pi);

    
    // write time to file
    FILE *fp;
    fp = fopen("piCUDA.csv", "w");
    fprintf(fp, "%f\n", diff);
    fclose(fp);

    free(randPoints);
    hipFree(dev_randPoints);
    hipFree(dev_inside);
    hipFree(states);

    exit(0);
}

__global__ void mcMethod(int iter, float *randPoints, int* inside){
    // get every other index
    int i = (threadIdx.x + blockDim.x * blockIdx.x)*2;
    int size = iter*2;

    // if the index is in bounds, calculate distance and use atomicAdd
    if(i < size-1){
        float distance = randPoints[i]*randPoints[i] + randPoints[i+1]*randPoints[i+1];
        if (distance <= 1){
            atomicAdd(&inside[0], 1);
        }
    }
}

__global__ void generateRandomNumbers(hiprandState_t* states, float* numbers, int n) {
    int i = (threadIdx.x + blockDim.x * blockIdx.x)*2; 

    // Generate a random number and store it in the numbers array
    if(i < n-1){
        // initialize curand (seed, sequence, offset, state)
        hiprand_init(1234, i, 0, &states[i]);
        hiprand_init(1234, i+1, 0, &states[i+1]);

        numbers[i] = hiprand_uniform(&states[i]);
        numbers[i+1] = hiprand_uniform(&states[i+1]);
        //printf("%d (%f, %f)\n", i, numbers[i], numbers[i+1]);
    }
}
